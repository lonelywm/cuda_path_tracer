#include "hip/hip_runtime.h"
#include <pch.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "../Triangle.hpp"
#include "../BVH.h"


__device__
int findSplit(unsigned int* sortedMortonCodes,
    int first, int last
) {
    // Identical Morton codes => split the range in the middle.
    unsigned int firstCode = sortedMortonCodes[first];
    unsigned int lastCode = sortedMortonCodes[last];

    if (firstCode == lastCode)
        return (first + last) >> 1;

    // Calculate the number of highest bits that are the same
    // for all objects, using the count-leading-zeros intrinsic.

    int commonPrefix = __clz(firstCode ^ lastCode);

    // Use binary search to find where the next bit differs.
    // Specifically, we are looking for the highest object that
    // shares more than commonPrefix bits with the first one.

    int split = first; // initial guess
    int step = last - first;

    do
    {
        step = (step + 1) >> 1; // exponential decrease
        int newSplit = split + step; // proposed new position

        if (newSplit < last)
        {
            unsigned int splitCode = sortedMortonCodes[newSplit];
            int splitPrefix = __clz(firstCode ^ splitCode);
            if (splitPrefix > commonPrefix)
                split = newSplit; // accept proposal
        }
    }
    while (step > 1);
    return split;
}

__device__
int2 determineRange(uint* sortedMortonCodes, int numTriangles, int idx)
{
    //determine the range of keys covered by each internal node (as well as its children)
    //direction is found by looking at the neighboring keys ki-1 , ki , ki+1
    //the index is either the beginning of the range or the end of the range
    int direction = 0;
    int common_prefix_with_left = 0;
    int common_prefix_with_right = 0;

    common_prefix_with_right = __clz(sortedMortonCodes[idx] ^ sortedMortonCodes[idx + 1]);
    if(idx == 0){
        common_prefix_with_left = -1;
    }
    else
    {
        common_prefix_with_left = __clz(sortedMortonCodes[idx] ^ sortedMortonCodes[idx - 1]);

    }

    direction = ( (common_prefix_with_right - common_prefix_with_left) > 0 ) ? 1 : -1;
    int min_prefix_range = 0;

    if(idx == 0)
    {
        min_prefix_range = -1;

    }
    else
    {
        min_prefix_range = __clz(sortedMortonCodes[idx] ^ sortedMortonCodes[idx - direction]); 
    }

    int lmax = 2;
    int next_key = idx + lmax*direction;

    while((next_key >= 0) && (next_key <  numTriangles) && (__clz(sortedMortonCodes[idx] ^ sortedMortonCodes[next_key]) > min_prefix_range))
    {
        lmax *= 2;
        next_key = idx + lmax*direction;
    }
    //find the other end using binary search
    unsigned int l = 0;

    do
    {
        lmax = (lmax + 1) >> 1; // exponential decrease
        int new_val = idx + (l + lmax)*direction ; 

        if(new_val >= 0 && new_val < numTriangles )
        {
            unsigned int Code = sortedMortonCodes[new_val];
            int Prefix = __clz(sortedMortonCodes[idx] ^ Code);
            if (Prefix > min_prefix_range)
                l = l + lmax;
        }
    }
    while (lmax > 1);

    int j = idx + l*direction;

    int left = 0 ; 
    int right = 0;
    
    if(idx < j) {
        left = idx;
        right = j;
    } else {
        left = j;
        right = idx;
    }

    // printf("idx : (%d) returning range (%d, %d) \n" , idx , left, right);
    return make_int2(left,right);
}

__device__
unsigned int expandBits(unsigned int v) {
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__device__
unsigned int morton3D(float x, float y, float z) {
    x = min(max(x * 1024.0f, 0.0f), 1023.0f);
    y = min(max(y * 1024.0f, 0.0f), 1023.0f);
    z = min(max(z * 1024.0f, 0.0f), 1023.0f);
    unsigned int xx = expandBits((unsigned int)x);
    unsigned int yy = expandBits((unsigned int)y);
    unsigned int zz = expandBits((unsigned int)z);
    return xx * 4 + yy * 2 + zz;
}

// ===============================================================

__global__
void computeMortonCodesKernel(uint* mCodes, uint* objIds, BoundingBox* boxs, int numTri, Vec3f min, Vec3f max) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numTri) return;

    objIds[idx] = idx;
    Vec3f centroid = boxs[idx].getCentroid();
    centroid.x = (centroid.x - min.x)/(max.x - min.x);
    centroid.y = (centroid.y - min.y)/(max.y - min.y);
    centroid.z = (centroid.z - min.z)/(max.z - min.z);
    mCodes[idx] = morton3D(centroid.x, centroid.y, centroid.z);

    // printf("Max(%f, %f, %f), Min(%f, %f, %f)\n", max.x, max.y, max.z, min.x, min.y, min.z);
    // printf("BMax(%f, %f, %f), BMin(%f, %f, %f)\n", boxs[idx].Max.x, boxs[idx].Max.y, boxs[idx].Max.z,
    //     boxs[idx].Min.x, boxs[idx].Min.y, boxs[idx].Min.z);
    // printf("x(%f), y(%f), z(%f), [%08x]\n", centroid.x, centroid.y, centroid.z, mCodes[idx]);
}


__global__ 
void generateHierarchyKernel(
    uint* sortedMortonCodes, uint* sorted_object_ids, BVHNode* internalNodes,
    BVHNode* leafNodes, int numTri, BoundingBox* BBoxs
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numTri - 1) return;

    internalNodes[idx].IsLeaf = false;

    int2 range = determineRange(sortedMortonCodes, numTri, idx);
    int first = range.x;
    int last = range.y;

    int split = findSplit(sortedMortonCodes, first, last);
    BVHNode* childA;

    bool isLeafA = false;
    bool isLeafB = false;
    if (split == first) {
        childA = &leafNodes[split];
        isLeafA = true;
    }
    else {
        childA = &internalNodes[split];
    }

    BVHNode* childB;
    if (split + 1 == last) {
        childB = &leafNodes[split + 1];
        isLeafB = true;
    } else {
        childB = &internalNodes[split + 1];
    }
    
    // printf("idx: %03d spl: %03d CA(%02d) Leaf(%d) CB(%02d) Leaf(%d)\n", idx, split, split, isLeafA, split+1, isLeafB);

    // printf("LeafA %02d Parent: %d (%d) \n", split, idx,   &internalNodes[idx]);
    // printf("LeafB %02d Parent: %d (%d) \n", split+1, idx, &internalNodes[idx]);

    childA->Parent = &internalNodes[idx];
    childB->Parent = &internalNodes[idx];
    internalNodes[idx].ChildA = childA;
    internalNodes[idx].ChildB = childB;

    // printf("split: %d", idx);
    // printf("PA %d, ", childA->Parent);
    // printf("PB %d, ", childB->Parent);
    // printf("P %d     ", internalNodes[idx]);
}

__global__
void infoBBoxesKernel(BVHNode* leafNodes, BVHNode* internalNodes, int numTriangles) {
    BVHNode* nodes = leafNodes;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numTriangles) return;
    
    BVHNode* node = &nodes[idx];
    // printf("Leaf(%d) ObjId(%d), min(%0.4f, %0.4f, %0.4f) max(%0.4f, %0.4f, %0.4f) \n", 
    //     idx, node->ObjectId, node->BBox.Min.x, node->BBox.Min.y, node->BBox.Min.z, node->BBox.Max.x, node->BBox.Max.y, node->BBox.Max.z
    // );
}

__global__ 
void computeBBoxesKernel(BVHNode* leafNodes, BVHNode* internalNodes, int numTriangles)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numTriangles)
        return;

    BVHNode* Parent = leafNodes[idx].Parent;

    // if (idx < numTriangles) {
    //     printf("leaf: %d\n", leafNodes);
    //     printf("inter: %d\n", internalNodes);
    //     printf("Parent: %d\n", leafNodes[idx].Parent);
    //     printf("ChildA: %d\n", leafNodes[idx].Parent->ChildA);
    //     printf("ChildB: %d\n", leafNodes[idx].Parent->ChildB);
    // }

    // printf("%d", leafNodes[idx].BBox.isEmpty());
    // printf("ChildAB (%d)\n", leafNodes[idx].Parent->ChildA->BBox.isEmpty());

    while(Parent)
    {
        if(!Parent->ChildA->BBox.isEmpty() && !Parent->ChildB->BBox.isEmpty())
        {
            // Parent->BBox.bEmpty = true;
            Parent->BBox.merge(Parent->ChildA->BBox);
            Parent->BBox.merge(Parent->ChildB->BBox);
            // printf("**********parent child relationships**********\n");
            // printf(
            //     "parent idx (%d), min(%0.4f, %0.4f, %0.4f) max(%0.4f, %0.4f, %0.4f) \n"
            //     "childA leaf(%d) min(%0.4f, %0.4f, %0.4f) max(%0.4f, %0.4f, %0.4f) \n"
            //     "childB leaf(%d) min(%0.4f, %0.4f, %0.4f) max(%0.4f, %0.4f, %0.4f) \n\n",
            //     Parent - internalNodes, Parent->BBox.Min.x, Parent->BBox.Min.y, Parent->BBox.Min.z, Parent->BBox.Max.x, Parent->BBox.Max.y, Parent->BBox.Max.z,
            //     Parent->ChildA->IsLeaf, Parent->ChildA->BBox.Min.x, Parent->ChildA->BBox.Min.y, Parent->ChildA->BBox.Min.z, Parent->ChildA->BBox.Max.x, Parent->ChildA->BBox.Max.y, Parent->ChildA->BBox.Max.z,
            //     Parent->ChildB->IsLeaf, Parent->ChildB->BBox.Min.x, Parent->ChildB->BBox.Min.y, Parent->ChildB->BBox.Min.z, Parent->ChildB->BBox.Max.x, Parent->ChildB->BBox.Max.y, Parent->ChildB->BBox.Max.z
            // );
            Parent = Parent->Parent;
        } else{
            // printf(
            //     "skip (%d), (%d), (%d) %d \n"
            //     "min(%0.4f, %0.4f, %0.4f) max(%0.4f, %0.4f, %0.4f) \n\n",
            //     Parent - internalNodes, Parent->ChildA->BBox.isEmpty(), Parent->ChildB->BBox.isEmpty(), Parent->ChildA,
            //     Parent->ChildA->BBox.Min.x, Parent->ChildA->BBox.Min.y, Parent->ChildA->BBox.Min.z, 
            //     Parent->ChildB->BBox.Min.x, Parent->ChildB->BBox.Min.y, Parent->ChildB->BBox.Min.z
            // );
            break;
        }
        // __syncthreads();
    }
}

__global__ 
void setupLeafNodesKernel(uint* sorted_object_ids, BVHNode* leafNodes, BoundingBox* bboxes, int numTri) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numTri) return;
    int ObjectId = leafNodes[idx].ObjectId = sorted_object_ids[idx];
    leafNodes[idx].ChildA = nullptr;
    leafNodes[idx].ChildB = nullptr;
    leafNodes[idx].IsLeaf = true;
    leafNodes[idx].BBox   = bboxes[ObjectId];
}

__global__ 
void printfMortonCodes(uint* mortonCodes, int count) {
    for (int i=0; i<count; i++) {
        // for (int j=0; j<32; j++) {
        //     if (((int)mortonCodes[i] << j) & 1) {
        //         printf("1");
        //     } else {
        //         printf("0");
        //     }
        // }
        // printf("\n");
        // printf("%x\n", mortonCodes[i]);
    }
}

// BVH ==============================================================
void BVH::setup(
    Point* pts, uint* indices, BoundingBox* mBBoxs, int numTriangles, Vec3f min, Vec3f max
) {
    _pts = pts;
    _indices = indices;
    _bboxs = mBBoxs;

    hipMalloc(&_mortonCodes,   numTriangles*sizeof(unsigned int));
    hipMalloc(&_objectIds,     numTriangles*sizeof(unsigned int));
    hipMalloc(&_leafNodes,     numTriangles*sizeof(BVHNode));
    hipMalloc(&_internalNodes, (numTriangles - 1)*sizeof(BVHNode));

    int threadsPerBlock = 256;
    int blocksPerGrid = (numTriangles + threadsPerBlock - 1) / threadsPerBlock;

    // 1.comput morton codes
    computeMortonCodesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        _mortonCodes, _objectIds, _bboxs, numTriangles, min, max
    );

    // printf("min %4f, max: %4f", min.x, min.y, min.z);

    // 2. sort morton codes
    thrust::device_ptr<unsigned int> dev_mortonCodes(_mortonCodes);
    thrust::device_ptr<unsigned int> dev_object_ids(_objectIds);
    thrust::sort_by_key(dev_mortonCodes, dev_mortonCodes + numTriangles, dev_object_ids);

    // 3. build tree
    setupLeafNodesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        _objectIds, _leafNodes, _bboxs, numTriangles
    );
    generateHierarchyKernel<<<blocksPerGrid, threadsPerBlock>>>(
        _mortonCodes, _objectIds, _internalNodes, _leafNodes, numTriangles, _bboxs
    );

    printfMortonCodes<<<1, 1>>>(_mortonCodes, numTriangles);

    infoBBoxesKernel<<<blocksPerGrid, threadsPerBlock>>>(_leafNodes, _internalNodes, numTriangles);

    computeBBoxesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        _leafNodes, _internalNodes, numTriangles
    );
}

BVH::~BVH(){
    hipFree(_mortonCodes);
    hipFree(_objectIds);
    hipFree(_leafNodes);
    hipFree(_internalNodes);
}


